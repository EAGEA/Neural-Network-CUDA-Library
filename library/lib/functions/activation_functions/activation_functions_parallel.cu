#include "hip/hip_runtime.h"
//
// Created by Emilien Aufauvre on 12/12/2021.
//

#include "activation_functions.h"


using namespace cudaNN;


/**
 * Kernel functions.
 */


__global__ void __kernel_linear(float *results, float *inputs,
                                size_t nb_rows, size_t nb_cols)
{
    size_t col = blockIdx.x * blockDim.x + threadIdx.x;
    size_t row = blockIdx.y * blockDim.y + threadIdx.y;
    size_t index = row * nb_cols + col;

    // Check if the thread is in the matrix dimensions.
    if (row < nb_rows && col < nb_cols)
    {
        results[index] = inputs[index];
    }
}

__global__ void __kernel_linear_derivative(float *results, float *inputs,
                                           size_t nb_rows, size_t nb_cols)
{
    size_t col = blockIdx.x * blockDim.x + threadIdx.x;
    size_t row = blockIdx.y * blockDim.y + threadIdx.y;
    size_t index = row * nb_cols + col;

    // Check if the thread is in the matrix dimensions.
    if (row < nb_rows && col < nb_cols)
    {
        results[index] = 1.f;
    }
}

__global__ void __kernel_binary_step(float *results, float *inputs,
                                     size_t nb_rows, size_t nb_cols)
{
    size_t col = blockIdx.x * blockDim.x + threadIdx.x;
    size_t row = blockIdx.y * blockDim.y + threadIdx.y;
    size_t index = row * nb_cols + col;

    // Check if the thread is in the matrix dimensions.
    if (row < nb_rows && col < nb_cols)
    {
        results[index] = inputs[index] < 0.f ? 0.f : 1.f;
    }
}

__global__ void __kernel_binary_step_derivative(float *results, float *inputs,
                                                size_t nb_rows, size_t nb_cols)
{
    size_t col = blockIdx.x * blockDim.x + threadIdx.x;
    size_t row = blockIdx.y * blockDim.y + threadIdx.y;
    size_t index = row * nb_cols + col;

    // Check if the thread is in the matrix dimensions.
    if (row < nb_rows && col < nb_cols)
    {
        results[index] = 0.f;
    }
}

__global__ void __kernel_sigmoid(float *results, float *inputs,
                                 size_t nb_rows, size_t nb_cols)
{
    size_t col = blockIdx.x * blockDim.x + threadIdx.x;
    size_t row = blockIdx.y * blockDim.y + threadIdx.y;
    size_t index = row * nb_cols + col;

    // Check if the thread is in the matrix dimensions.
    if (row < nb_rows && col < nb_cols)
    {
        results[index] = 1.f / (1.f + expf(-inputs[index]));
    }
}

__global__ void __kernel_sigmoid_derivative(float *results, float *inputs,
                                            size_t nb_rows, size_t nb_cols)
{
    size_t col = blockIdx.x * blockDim.x + threadIdx.x;
    size_t row = blockIdx.y * blockDim.y + threadIdx.y;
    size_t index = row * nb_cols + col;

    // Check if the thread is in the matrix dimensions.
    if (row < nb_rows && col < nb_cols)
    {
        float sigmoid = 1.f / (1.f + expf(-inputs[index]));
        results[index] = sigmoid * (1.f - sigmoid);
    }
}

__global__ void __kernel_relu(float *results, float *inputs,
                              size_t nb_rows, size_t nb_cols)
{
    size_t col = blockIdx.x * blockDim.x + threadIdx.x;
    size_t row = blockIdx.y * blockDim.y + threadIdx.y;
    size_t index = row * nb_cols + col;

    // Check if the thread is in the matrix dimensions.
    if (row < nb_rows && col < nb_cols)
    {
        results[index] = fmax(0.f, inputs[index]);
    }
}

__global__ void __kernel_relu_derivative(float *results, float *inputs,
                                         size_t nb_rows, size_t nb_cols)
{
    size_t col = blockIdx.x * blockDim.x + threadIdx.x;
    size_t row = blockIdx.y * blockDim.y + threadIdx.y;
    size_t index = row * nb_cols + col;

    // Check if the thread is in the matrix dimensions.
    if (row < nb_rows && col < nb_cols)
    {
        results[index] = inputs[index] > 0 ? 1.f : 0.f;
    }
}

__global__ void __kernel_tanh(float *results, float *inputs,
                              size_t nb_rows, size_t nb_cols)
{
    size_t col = blockIdx.x * blockDim.x + threadIdx.x;
    size_t row = blockIdx.y * blockDim.y + threadIdx.y;
    size_t index = row * nb_cols + col;

    // Check if the thread is in the matrix dimensions.
    if (row < nb_rows && col < nb_cols)
    {
        results[index] = tanhf(inputs[index]);
    }
}

__global__ void __kernel_tanh_derivative(float *results, float *inputs,
                                         size_t nb_rows, size_t nb_cols)
{
    size_t col = blockIdx.x * blockDim.x + threadIdx.x;
    size_t row = blockIdx.y * blockDim.y + threadIdx.y;
    size_t index = row * nb_cols + col;

    // Check if the thread is in the matrix dimensions.
    if (row < nb_rows && col < nb_cols)
    {
        float tanh_ = tanhf(inputs[index]);
        results[index] = 1.f - tanh_ * tanh_;
    }
}

__global__ void __kernel_softmax(float *results, float *inputs,
                                 float *sum,
                                 size_t nb_rows, size_t nb_cols)
{
    // Do a reduction to compute the sum.
    extern __shared__ float shared_sum[];
    // Copy into shared memory.
    shared_sum[threadIdx.x] = expf(inputs[blockIdx.x * blockDim.x + threadIdx.x]);

    __syncthreads();

    for (size_t stride = blockDim.x / 2; stride > 0; stride >>= 1)
    {
        if (threadIdx.x < stride)
        {
            shared_sum[threadIdx.x] += shared_sum[threadIdx.x + stride];
        }
    }

    __syncthreads();

    // Retrieve and sum the sum computed by each block.
    if (threadIdx.x == 0)
    {
        atomicAdd(sum, shared_sum[0]);
    }

    // Compute the softmax using the previously computed sum.
    size_t index = blockIdx.x * blockDim.x + threadIdx.x;

    __syncthreads();

    if (index < nb_rows * nb_cols)
    {
        results[index] = expf(inputs[index]) / sum[0];
    }
}

__global__ void __kernel_softmax_derivative(float *results, float *inputs,
                                            float *sum,
                                            size_t nb_rows, size_t nb_cols)
{
    // Do a reduction to compute the sum.
    extern __shared__ float shared_sum[];
    // Copy into shared memory.
    shared_sum[threadIdx.x] = inputs[blockIdx.x * blockDim.x + threadIdx.x];

    __syncthreads();

    for (size_t stride = blockDim.x / 2; stride > 0; stride >>= 1)
    {
        if (threadIdx.x < stride)
        {
            shared_sum[threadIdx.x] += shared_sum[threadIdx.x + stride];
        }
    }

    __syncthreads();

    // Retrieve and sum the sum computed by each block.
    if (threadIdx.x == 0)
    {
        atomicAdd(sum, shared_sum[0]);
    }

    // Compute the derivative using the previously computed sum.
    size_t index = blockIdx.x * blockDim.x + threadIdx.x;

    __syncthreads();

    if (index < nb_rows * nb_cols)
    {
        size_t row = index / nb_cols;
        size_t col = index % nb_cols;
        float softmax_x = expf(inputs[row]) / sum[0];
        float softmax_y = expf(inputs[col]) / sum[0];

        if (row == col)
        {
            results[index] = softmax_x * (1 - softmax_x);
        }
        else
        {
            results[index] = -softmax_x * softmax_y;
        }
    }
}

void __helper(const matrix &results, const matrix &inputs,
              void (kernel)(float *result, float *inputs, size_t nb_rows, size_t nb_cols))
{
    auto cuda_dims = util::get_cuda_2dims(inputs.get_dimensions());
    auto block_dims = cuda_dims.first;
    auto thread_dims = cuda_dims.second;

    float *device_data1;
    float *device_data2;

    // Prepare data on device.
    matrix_parallel::start_operation(results, &device_data1);
    matrix_parallel::start_operation(inputs, &device_data2);

    // Do computations with CUDA threads.
    kernel<<<block_dims, thread_dims>>>(
            device_data1, device_data2,
            results.get_dimensions().first, results.get_dimensions().second);
    // Wait for all threads.
    CUDA_CHECK(hipDeviceSynchronize());
    // Retrieve/free data from device.
    matrix_parallel::end_operation(results, &device_data1);
    matrix_parallel::end_operation(inputs, &device_data2);
}

void __helper_softmax(const matrix &results, const matrix &inputs,
                      void (kernel)(float *result, float *inputs, float *sum,
                                    size_t nb_rows, size_t nb_cols))
{
    // We use a reduction that assumes that the data is contained
    // in an array of size 2^n. Therefore, we round up to the next
    // power of 2 our matrix array.
    auto ceil2 = util::ceil2(inputs.get_length());
    auto cuda_dims = util::get_cuda_1dims(
            std::pair<size_t, size_t>(ceil2, 1));
    auto block_dims = cuda_dims.first;
    auto thread_dims = cuda_dims.second;

    // Sum init for softmax.
    float *device_data1;
    float *device_data2;
    float *sum;
    float zero = 0.f;

    // Prepare data on device.
    matrix_parallel::start_operation(results, &device_data1);
    // For the matrix on which we will do the reduction:
    // - Allocate memory on device (of size 2^n).
    CUDA_CHECK(hipMalloc(&device_data2, ceil2 * sizeof(float)));
    // - Copy the matrix to this memory.
    CUDA_CHECK(hipMemcpy(device_data2, inputs.get_data(),
                          inputs.get_length() * sizeof(float),
                          hipMemcpyHostToDevice));
    // - Allocate for the sum.
    CUDA_CHECK(hipMalloc(&sum, sizeof(float)));
    CUDA_CHECK(hipMemcpy(sum, &zero,
                          sizeof(float),
                          hipMemcpyHostToDevice));
    // Do computations with CUDA threads.
    kernel<<<block_dims, thread_dims, ceil2 * sizeof(float)>>>(
            device_data1, device_data2,
            sum,
            results.get_dimensions().first, results.get_dimensions().second);
    // Wait for all threads.
    CUDA_CHECK(hipDeviceSynchronize());
    // Retrieve/free data from device.
    matrix_parallel::end_operation(results, &device_data1);
    matrix_parallel::end_operation(inputs, &device_data2);
    CUDA_CHECK(hipFree(sum));
}


/**
 * Wrappers for call on host.
 */


void activation_functions_parallel::linear(std::vector<matrix *> m)
{
    __helper(*m[0], *m[1],__kernel_linear);
}

void activation_functions_parallel::linear_derivative(std::vector<matrix *> m)
{
    __helper(*m[0], *m[1],__kernel_linear_derivative);
}

void activation_functions_parallel::binary_step(std::vector<matrix *> m)
{
    __helper(*m[0], *m[1],__kernel_binary_step);
}

void activation_functions_parallel::binary_step_derivative(std::vector<matrix *> m)
{
    __helper(*m[0], *m[1],__kernel_binary_step_derivative);
}

void activation_functions_parallel::sigmoid(std::vector<matrix *> m)
{
    __helper(*m[0], *m[1],__kernel_sigmoid);
}

void activation_functions_parallel::sigmoid_derivative(std::vector<matrix *> m)
{
    __helper(*m[0], *m[1],__kernel_sigmoid_derivative);
}

void activation_functions_parallel::relu(std::vector<matrix *> m)
{
    __helper(*m[0], *m[1],__kernel_relu);
}

void activation_functions_parallel::relu_derivative(std::vector<matrix *> m)
{
    __helper(*m[0], *m[1],__kernel_relu_derivative);
}

void activation_functions_parallel::tanh(std::vector<matrix *> m)
{
    __helper(*m[0], *m[1],__kernel_tanh);
}

void activation_functions_parallel::tanh_derivative(std::vector<matrix *> m)
{
    __helper(*m[0], *m[1],__kernel_tanh_derivative);
}

void activation_functions_parallel::softmax(std::vector<matrix *> m)
{
    __helper_softmax(*m[0], *m[1],__kernel_softmax);
}

void activation_functions_parallel::softmax_derivative(std::vector<matrix *> m)
{
    __helper_softmax(*m[0], *m[1],__kernel_softmax_derivative);
}
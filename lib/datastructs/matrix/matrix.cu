#include "hip/hip_runtime.h"
//
// Created by Emilien Aufauvre on 09/12/2021.
//

#include "matrix.h"


using namespace cudaNN;


/**
 * Kernel functions.
 */


__global__ void __kernel_add(float *data1, const float *data2,
                             size_t nb_rows, size_t nb_cols)
{
    size_t col = blockIdx.x * blockDim.x + threadIdx.x;
    size_t row = blockIdx.y * blockDim.y + threadIdx.y;

    // Check if thread index is in the output dimensions.
    if (row < nb_rows && col < nb_cols)
    {
        data1[row * nb_cols + col] += data2[row * nb_cols + col];
    }
}

__global__ void __kernel_multiply(float *result,
                                  const float *data1, const float *data2,
                                  size_t nb_rows_1, size_t nb_cols_1,
                                  size_t nb_rows_2, size_t nb_cols_2)
{
    size_t col = blockIdx.x * blockDim.x + threadIdx.x;
    size_t row = blockIdx.y * blockDim.y + threadIdx.y;

    // Check if thread index is in the output dimensions.
    if (row < nb_rows_1 && col < nb_cols_2)
    {
        float sum = .0f;

        for (size_t i = 0; i < nb_cols_1; i ++)
        {
            sum += data1[row * nb_cols_1 + i] * data2[i * nb_cols_2 + col];
        }

        result[row * nb_cols_2 + col] = sum;
    }
}


/**
 * Wrappers for call on host.
 */


void matrix_cuda::start_operation(const matrix &m, float **device_data)
{
    // Allocate memory on device.
    CUDA_CHECK(hipMalloc(device_data, m.get_length() * sizeof(float)));
    // Copy the matrix to this memory.
    CUDA_CHECK(hipMemcpy(*device_data, m.get_data(),
                          m.get_length() * sizeof(float),
                          hipMemcpyHostToDevice));
}

void matrix_cuda::end_operation(const matrix &m, float **device_data)
{
    // Retrieve data from the device to the host (matrix).
    CUDA_CHECK(hipMemcpy(m.get_data(), *device_data,
                          m.get_length() * sizeof(float),
                          hipMemcpyDeviceToHost));
    // Free device memory.
    CUDA_CHECK(hipFree(*device_data));
}

void matrix_cuda::add(const dim3 &block_dims, const dim3 &thread_dims,
                      const matrix &m1, const matrix &m2)
{
    float *device_data1;
    float *device_data2;

    // Prepare data on device.
    start_operation(m1, &device_data1);
    start_operation(m2, &device_data2);
    // Do computations with CUDA threads.
    __kernel_add<<<block_dims, thread_dims>>>(
            device_data1, device_data2,
            m1.get_dimensions().first, m1.get_dimensions().second);
    // Wait for all threads.
    CUDA_CHECK(hipDeviceSynchronize());
    // Retrieve/free data from device.
    end_operation(m1, &device_data1);
    end_operation(m2, &device_data2);
}

void matrix_cuda::multiply(const dim3 &block_dims, const dim3 &thread_dims,
                           const matrix &m,
                           const matrix &m1, const matrix &m2)
{
    float *device_result;
    float *device_data1;
    float *device_data2;

    // Prepare data on device.
    start_operation(m, &device_result);
    start_operation(m1, &device_data1);
    start_operation(m2, &device_data2);
    // Do computations with CUDA threads.
    __kernel_multiply<<<block_dims, thread_dims>>>(
            device_result,
            device_data1, device_data2,
            m1.get_dimensions().first, m1.get_dimensions().second,
            m2.get_dimensions().first, m2.get_dimensions().second);
    // Wait for all threads.
    CUDA_CHECK(hipDeviceSynchronize());
    // Retrieve/free data from device.
    end_operation(m, &device_result);
    end_operation(m1, &device_data1);
    end_operation(m2, &device_data2);
}
#include "hip/hip_runtime.h"
//
// Created by Emilien Aufauvre on 09/12/2021.
//

#include "matrix.h"


using namespace cudaNN;


/**
 * Kernel functions.
 */


__global__ void __kernel_add(float *data1, const float *data2,
                             size_t nb_rows, size_t nb_cols)
{
    size_t col = blockIdx.x * blockDim.x + threadIdx.x;
    size_t row = blockIdx.y * blockDim.y + threadIdx.y;
    size_t index = row * nb_cols + col;

    // Check if thread index is in the output dimensions.
    if (row < nb_rows && col < nb_cols)
    {
        data1[index] += data2[index];
    }
}

__global__ void __kernel_multiply(float *result,
                                  const float *data1, const float *data2,
                                  size_t nb_rows_1, size_t nb_cols_1,
                                  size_t nb_rows_2, size_t nb_cols_2)
{
    size_t col = blockIdx.x * blockDim.x + threadIdx.x;
    size_t row = blockIdx.y * blockDim.y + threadIdx.y;

    // Check if thread index is in the output dimensions.
    if (row < nb_rows_1 && col < nb_cols_2)
    {
        float sum = .0f;

        for (size_t i = 0; i < nb_cols_1; i ++)
        {
            sum += data1[row * nb_cols_1 + i] * data2[i * nb_cols_2 + col];
        }

        result[row * nb_cols_2 + col] = sum;
    }
}

__global__ void __kernel_sum(float *data, size_t nb_rows, size_t nb_cols)
{
    __shared__ size_t length;

    // Perform a reduction on "data".
    size_t col = blockIdx.x * blockDim.x + threadIdx.x;
    size_t row = blockIdx.y * blockDim.y + threadIdx.y;
    size_t index = row * nb_cols + col;
    length = nb_rows * nb_cols;

    for (size_t size = length; size >= 1; size /= 2)
    {
        if (size != length && index < size)
        {
            data[index] += data[index + size];
        }

        __syncthreads();

        if (size % 2 != 0 && index == size - 1)
        {
            data[1] += data[index];
            size --;
        }
    }
}


/**
 * Wrappers for call on host.
 */


void matrix_cuda::start_operation(const matrix &m, float **device_data)
{
    // Allocate memory on device.
    CUDA_CHECK(hipMalloc(device_data, m.get_length() * sizeof(float)));
    // Copy the matrix to this memory.
    CUDA_CHECK(hipMemcpy(*device_data, m.get_data(),
                          m.get_length() * sizeof(float),
                          hipMemcpyHostToDevice));
}

void matrix_cuda::end_operation(const matrix &m, float **device_data)
{
    // Retrieve data from the device to the host (matrix).
    CUDA_CHECK(hipMemcpy(m.get_data(), *device_data,
                          m.get_length() * sizeof(float),
                          hipMemcpyDeviceToHost));
    // Free device memory.
    CUDA_CHECK(hipFree(*device_data));
}

void matrix_cuda::add(const dim3 &block_dims, const dim3 &thread_dims,
                      const matrix &m1, const matrix &m2)
{
    float *device_data1;
    float *device_data2;

    // Prepare data on device.
    start_operation(m1, &device_data1);
    start_operation(m2, &device_data2);
    // Do computations with CUDA threads.
    __kernel_add<<<block_dims, thread_dims>>>(
            device_data1, device_data2,
            m1.get_dimensions().first, m1.get_dimensions().second);
    // Wait for all threads.
    CUDA_CHECK(hipDeviceSynchronize());
    // Retrieve/free data from device.
    end_operation(m1, &device_data1);
    end_operation(m2, &device_data2);
}

void matrix_cuda::multiply(const dim3 &block_dims, const dim3 &thread_dims,
                           const matrix &m,
                           const matrix &m1, const matrix &m2)
{
    float *device_result;
    float *device_data1;
    float *device_data2;

    // Prepare data on device.
    start_operation(m, &device_result);
    start_operation(m1, &device_data1);
    start_operation(m2, &device_data2);
    // Do computations with CUDA threads.
    __kernel_multiply<<<block_dims, thread_dims>>>(
            device_result,
            device_data1, device_data2,
            m1.get_dimensions().first, m1.get_dimensions().second,
            m2.get_dimensions().first, m2.get_dimensions().second);
    // Wait for all threads.
    CUDA_CHECK(hipDeviceSynchronize());
    // Retrieve/free data from device.
    end_operation(m, &device_result);
    end_operation(m1, &device_data1);
    end_operation(m2, &device_data2);
}

void matrix_cuda::sum(const dim3 &block_dims, const dim3 &thread_dims,
                      float *result, const matrix &m)
{
    float *device_data;
    // Prepare data on device.
    start_operation(m, &device_data);
    // Do computations with CUDA threads.
    __kernel_sum<<<block_dims, thread_dims, sizeof(size_t)>>>(
            device_data,
            m.get_dimensions().first, m.get_dimensions().second);
    // Wait for all threads.
    CUDA_CHECK(hipDeviceSynchronize());
    // Retrieve/free data from device.
    CUDA_CHECK(hipMemcpy(result, device_data,
                          sizeof(float),
                          hipMemcpyDeviceToHost));
    CUDA_CHECK(hipFree(device_data));
}